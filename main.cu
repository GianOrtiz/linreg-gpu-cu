#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define cudaCheck(ans)                     \
  {                                        \
    gpu_assert((ans), __FILE__, __LINE__); \
  }
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}
#define BLOCK_SIZE 16

__global__ void kernel_matrix_mult(float *out, float *a, float *b, int n)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float tmp_sum = 0;

  if (row < n && col < n)
  {
    // each thread computes one element of the block sub-matrix
    for (int i = 0; i < n; i++)
    {
      tmp_sum += a[row * n + i] * b[i * n + col];
    }
    out[row * n + col] = tmp_sum;
  }
}

// Set up(and cleanup) for the matrix multiplication on the GPU
void gpu_matrix_mult(float *out, float *a, float *b, int N)
{
  int SIZE = N * N;
  float *d_a, *d_b, *d_out;

  // Allocate arrays in device memory
  cudaCheck(hipMalloc((void **)&d_a, sizeof(float) * SIZE));
  cudaCheck(hipMalloc((void **)&d_b, sizeof(float) * SIZE));
  cudaCheck(hipMalloc((void **)&d_out, sizeof(float) * SIZE));

  // Copy data from the host memory to the device memory
  cudaCheck(hipMemcpy(d_a, a, sizeof(float) * SIZE, hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_b, b, sizeof(float) * SIZE, hipMemcpyHostToDevice));

  dim3 threads_per_block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 blocks_per_grid(N / BLOCK_SIZE, N / BLOCK_SIZE);

  printf("using %d threads per block\n", threads_per_block.x * threads_per_block.y);
  printf("using %d blocks per grid\n", blocks_per_grid.x * blocks_per_grid.y);

  kernel_matrix_mult<<<blocks_per_grid, threads_per_block>>>(d_out, d_a, d_b, N);
  cudaCheck(hipPeekAtLastError());
  cudaCheck(hipDeviceSynchronize());

  // Copy result from device memory to the host memory
  cudaCheck(hipMemcpy(out, d_out, sizeof(float) * SIZE, hipMemcpyDeviceToHost));
  cudaCheck(hipDeviceSynchronize());

  // Free arrays in device memory
  cudaCheck(hipFree(d_a));
  cudaCheck(hipFree(d_b));
  cudaCheck(hipFree(d_out));
}

void inspect_gpu()
{
  int device;
  hipGetDevice(&device);
  struct hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, device);
  printf("---- GPU INFO -------\n");
  printf("\tusing %d multiprocessors\n", properties.multiProcessorCount);
  printf("\tmax blocks per processor: %d\n", properties.maxBlocksPerMultiProcessor);
  printf("\tmax threads per block: %d\n", properties.maxThreadsPerBlock);
  printf("\tmax threads per processor: %d\n\n", properties.maxThreadsPerMultiProcessor);
}

void cpu_matrix_mult(float *out, float *a, float *b, int N)
{
  for (int y = 0; y < N; y++)
  {
    for (int x = 0; x < N; x++)
    {
      float sum = 0.f;
      for (int n = 0; n < N; n++)
      {
        sum += a[y * N + n] * b[n * N + x];
      }
      out[y * N + x] = sum;
    }
  }
}

double mean_squared_error(float *a, float *b, int N)
{
  double err = 0;
  for (int y = 0; y < N; y++)
  {
    for (int x = 0; x < N; x++)
    {
      int i = y * N + x;
      err += pow(a[i] - b[i], 2);
    }
  }
  return err;
}

int main()
{
  int N = 1024;
  int SIZE = N * N;

  float *a = (float *)malloc(sizeof(float) * SIZE);
  float *b = (float *)malloc(sizeof(float) * SIZE);

  // Initialize matrices on the host
  for (int i = 0; i < N; i++)
  {
    for (int j = 0; j < N; j++)
    {
      a[i * N + j] = sin(i);
      b[i * N + j] = cos(j);
    }
  }

  inspect_gpu();

  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  cudaCheck(hipEventCreate(&start));
  cudaCheck(hipEventCreate(&stop));

  float *out = (float *)malloc(sizeof(float) * SIZE);
  cudaCheck(hipEventRecord(start));
  cudaCheck(hipEventSynchronize(start));
  gpu_matrix_mult(out, a, b, N);
  cudaCheck(hipEventRecord(stop));
  cudaCheck(hipEventSynchronize(stop));
  float gpu_msec_total = 0.0f;
  cudaCheck(hipEventElapsedTime(&gpu_msec_total, start, stop));

  float *cpu_out = (float *)malloc(sizeof(float) * SIZE);
  cudaCheck(hipEventRecord(start));
  cudaCheck(hipEventSynchronize(start));
  cpu_matrix_mult(cpu_out, a, b, N);
  cudaCheck(hipEventRecord(stop));
  cudaCheck(hipEventSynchronize(stop));
  float cpu_msec_total = 0.0f;
  cudaCheck(hipEventElapsedTime(&cpu_msec_total, start, stop));

  double err = mean_squared_error(out, cpu_out, N);

  printf("Mean squared error: %f\n", err);
  printf("Time elapsed GPU: %.2fms. CPU: %.2fms\n", gpu_msec_total, cpu_msec_total);

  // Deallocate host memory
  free(a);
  free(b);
  free(out);
  free(cpu_out);
  cudaCheck(hipEventDestroy(start));
  cudaCheck(hipEventDestroy(stop));
}
