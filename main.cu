#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <queue>
#include <string>
#include <istream>
#include <ostream>
#include <fstream>
#include <iostream>
#include <thread>
#include <array>
#include <mutex>
#include <optional>

#define cudaCheck(ans)                     \
  {                                        \
    gpu_assert((ans), __FILE__, __LINE__); \
  }
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

template <class T, size_t TElemCount>
class circular_buffer
{
public:
  explicit circular_buffer() = default;

  void put(T item)
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    buf_[head_] = item;

    if (full_)
    {
      tail_ = (tail_ + 1) % TElemCount;
    }

    head_ = (head_ + 1) % TElemCount;

    full_ = head_ == tail_;
  }

  // __device__ void gpu_put(T *item)
  // {
  //   std::lock_guard<std::recursive_mutex> lock(mutex_);

  //   cudaCheck(cudaMemCpy(&buf_[head_], item, sizeof(T), cudaMemcpyHostToDevice));

  //   if (full_)
  //   {
  //     tail_ = (tail_ + 1) % TElemCount;
  //   }

  //   head_ = (head_ + 1) % TElemCount;

  //   full_ = head_ == tail_;
  // }

  T get()
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    // if (empty())
    // {
    //   return std::nullopt;
    // }

    // Read data and advance the tail (we now have a free space)
    auto val = buf_[tail_];
    full_ = false;
    tail_ = (tail_ + 1) % TElemCount;

    return val;
  }

  void reset()
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);
    head_ = tail_;
    full_ = false;
  }

  bool empty()
  {
    // Can have a race condition in a multi-threaded application
    std::lock_guard<std::recursive_mutex> lock(mutex_);
    // if head and tail are equal, we are empty
    return (!full_ && (head_ == tail_));
  }

  bool full()
  {
    // If tail is ahead the head by 1, we are full
    return full_;
  }

  size_t capacity()
  {
    return TElemCount;
  }

  size_t size()
  {
    // A lock is needed in size ot prevent a race condition, because head_, tail_, and full_
    // can be updated between executing lines within this function in a multi-threaded
    // application
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    size_t size = TElemCount;

    if (!full_)
    {
      if (head_ >= tail_)
      {
        size = head_ - tail_;
      }
      else
      {
        size = TElemCount + head_ - tail_;
      }
    }

    return size;
  }

private:
  mutable std::recursive_mutex mutex_;
  mutable std::array<T, TElemCount> buf_;
  mutable size_t head_ = 0;
  mutable size_t tail_ = 0;
  mutable bool full_ = 0;
};

const int N = 16;
const int SIZE = N * N;
const int WINDOW = 100;

const uint BUFFER_SIZE = 10000;
class Block
{
  float data[BUFFER_SIZE];
  uint index;

public:
  Block(float data[BUFFER_SIZE], uint index)
  {
    std::copy(data, data + BUFFER_SIZE, this->data);
    this->index = index;
  };
};

class Reader
{
  static const size_t BUFFER_COUNT = 100;

public:
  bool done = false;
  uint read_until = 0;
  circular_buffer<Block, Reader::BUFFER_COUNT> value_buffer;
  explicit Reader() = default;
  float *read(std::string filename)
  {
    std::ifstream in_file(filename, std::ifstream::binary);
    if (!in_file)
    {
      std::cerr << "Failed opening file" << std::endl;
      exit(1);
    }

    // Stop eating new lines in binary mode!!!
    in_file.unsetf(std::ios::skipws);

    while (!in_file.eof())
    {
      while (value_buffer.full())
      {
      }
      float *read_buffer = new float[BUFFER_SIZE];
      in_file.read((char *)read_buffer, BUFFER_SIZE * sizeof(float));
      value_buffer.put(Block(read_buffer, read_until));
      read_until += BUFFER_SIZE;
    }
    done = true;
    in_file.close();
  }
};

class DataScheduler
{
  static const size_t BUFFER_COUNT = 100;
  Reader *reader;
  uint processed_until = 0;
  // circular_buffer<Block, DataScheduler::BUFFER_COUNT> *device_buffer;

public:
  DataScheduler(Reader *reader)
  {
    this->reader = reader;
    // cudaCheck(cudaMalloc(&device_buffer, sizeof(circular_buffer<float, DataScheduler::BUFFER_COUNT>)));
  }

  void loop()
  {
    while (!reader->done)
    {
      while (reader->value_buffer.empty())
      {
      }

      auto block = reader->value_buffer.get();
      // device_buffer->gpu_put(&block);
    }
  }
};

int main()
{
  Reader reader();
}
