#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <queue>
#include <string>
#include <istream>
#include <ostream>
#include <fstream>
#include <iostream>
#include <thread>
#include <array>
#include <mutex>
#include <optional>
#include <ctime>
#include <chrono>

#define cudaCheck(ans)                     \
  {                                        \
    gpu_assert((ans), __FILE__, __LINE__); \
  }
inline void gpu_assert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s:%d\n", hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

#define KERNEL_BLOCK_SIZE 16

template <class T, size_t TElemCount>
class circular_buffer
{
public:
  explicit circular_buffer()
  {
    buf_ = {};
  }

  void put(T item)
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    buf_[head_] = item;

    if (full_)
    {
      tail_ = (tail_ + 1) % TElemCount;
    }

    head_ = (head_ + 1) % TElemCount;

    full_ = head_ == tail_;
  }

  T get()
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    // Read data and advance the tail (we now have a free space)
    auto val = buf_[tail_];
    full_ = false;
    tail_ = (tail_ + 1) % TElemCount;

    return val;
  }

  void reset()
  {
    std::lock_guard<std::recursive_mutex> lock(mutex_);
    head_ = tail_;
    full_ = false;
  }

  bool empty()
  {
    // Can have a race condition in a multi-threaded application
    std::lock_guard<std::recursive_mutex> lock(mutex_);
    // if head and tail are equal, we are empty
    return (!full_ && (head_ == tail_));
  }

  bool full()
  {
    // If tail is ahead the head by 1, we are full
    return full_;
  }

  size_t capacity()
  {
    return TElemCount;
  }

  size_t size()
  {
    // A lock is needed in size ot prevent a race condition, because head_, tail_, and full_
    // can be updated between executing lines within this function in a multi-threaded
    // application
    std::lock_guard<std::recursive_mutex> lock(mutex_);

    size_t size = TElemCount;

    if (!full_)
    {
      if (head_ >= tail_)
      {
        size = head_ - tail_;
      }
      else
      {
        size = TElemCount + head_ - tail_;
      }
    }

    return size;
  }

private:
  mutable std::recursive_mutex mutex_;
  mutable std::array<T, TElemCount> buf_;
  mutable size_t head_ = 0;
  mutable size_t tail_ = 0;
  mutable bool full_ = 0;
};

const int WINDOW = 100;

const uint BUFFER_SIZE = KERNEL_BLOCK_SIZE * 64;
const uint CHUNK_SIZE = BUFFER_SIZE + WINDOW;
class Chunk
{
public:
  uint index;
  Chunk() = default;
  float data[CHUNK_SIZE];
  Chunk(std::array<float, CHUNK_SIZE> data, uint index) : index(index)
  {
    std::copy(data.begin(), data.end(), this->data);
  };
};

#define SAMPLE_PERIOD_MS 10
// weight is the weight matrix
// weight is calculated as follows
// weight = (X^T * X)^-1 * X^T * Y
// where X is the input matrix, Y is the output matrix
// X is not-provided, as the samples are provided regularly in time
// so we can calculate the X from the index(index * SAMPLE_PERIOD_MS)
// X is n x 2
// Y is n x 2
// B(weight) is 2 x 2
// X^T is 2 x n
// (X^T * X) is 2 x 2
// (X^T * X)^-1 is 2 x 2
// (X^T * X)^-1 * X^T is 2 x n
__global__ void kernel_matrix_mult(Chunk *weight, Chunk *chunk)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (chunk->index == 0 && index < WINDOW)
  {
    return;
  }

  float tmp_sum = 0;

#define X(i) (SAMPLE_PERIOD_MS * i)

  float tmp_inv_matrix[4] = {0, 0, 0, 0}; // (X^T * X)
  float inv_matrix[4] = {0, 0, 0, 0};     // (X^T * X)^-1
  for (int i = WINDOW; i > 0; i--)
  {
    tmp_inv_matrix[0] += 1 * 1;
    tmp_inv_matrix[1] += X(i) * 1;
    tmp_inv_matrix[2] += 1 * X(i);
    tmp_inv_matrix[3] += X(i) * X(i);
  }
  // inverse of matrix
  // (1 / (a*d - b*c)) * [d, -b; -c, a]
  auto inv_det = 1 / (tmp_inv_matrix[0] * tmp_inv_matrix[3] - tmp_inv_matrix[1] * tmp_inv_matrix[2]);
  inv_matrix[0] = inv_det * tmp_inv_matrix[3];
  inv_matrix[1] = -inv_det * tmp_inv_matrix[1];
  inv_matrix[2] = -inv_det * tmp_inv_matrix[2];
  inv_matrix[3] = inv_det * tmp_inv_matrix[0];

  float inv_times_xt[WINDOW * 2] = {}; // (X^T * X)^-1 * X^T
  for (int i = WINDOW; i > 0; i--)
  {
    // [a, b] * [1   1 ...  1 ]
    // [c, b]   [x0 x1 ... xn ]
    inv_times_xt[i * 2] = inv_matrix[0] * 1 + inv_matrix[1] * X(i);
    inv_times_xt[i * 2 + 1] = inv_matrix[2] * 1 + inv_matrix[3] * X(i);
  }
  // each thread computes one element of the block sub-matrix
  for (int i = WINDOW; i > 0; i--)
  {
    auto Y = chunk->data;
    tmp_sum += inv_times_xt[i] * Y[i];
  }
  weight->data[index] = tmp_sum;
  weight->index = chunk->index;
}

// reads a one-dimensional CSV file
class Reader
{
  static const size_t BUFFER_COUNT = 100;

public:
  bool done = false;
  uint read_until = 0;
  circular_buffer<Chunk *, Reader::BUFFER_COUNT> value_buffer;
  Chunk *previous_chunk;
  explicit Reader(){};

  float *read(std::string filename)
  {
    std::ifstream in_file(filename, std::ifstream::binary);
    if (!in_file)
    {
      std::cerr << "Failed opening file" << std::endl;
      exit(1);
    }

    // Stop eating new lines in binary mode!!!
    in_file.unsetf(std::ios::skipws);

    auto start = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());
    int read_size = 0;

    std::string line{};
    // read the first line(column headers)
    std::getline(in_file, line);
    while (!in_file.eof())
    {
      while (value_buffer.full())
      {
      }
      std::array<float, BUFFER_SIZE> read_buffer;
      size_t i = 0;
      while (std::getline(in_file, line) && i < BUFFER_SIZE)
        read_buffer[i++] = std::stof(line);
      std::array<float, CHUNK_SIZE> chunk_read_buffer;
      std::copy(read_buffer.begin(), read_buffer.end(), &chunk_read_buffer.at(WINDOW));
      if (previous_chunk != nullptr)
      {
        std::copy(std::end(previous_chunk->data) - WINDOW, std::end(previous_chunk->data), chunk_read_buffer.begin());
      }

      Chunk *chunk = new Chunk(chunk_read_buffer, read_until);
      value_buffer.put(chunk);
      read_until += BUFFER_SIZE;
      previous_chunk = chunk;
      read_size += BUFFER_SIZE * sizeof(float);
    }
    auto end = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // read_size in bytes, time_diff in milliseconds
    float mean = ((float)read_size / (float)time_diff);
    std::printf("READER:\nTempo: %dms\nLido: %d bytes\nMédia: %f bytes/ms\n\n", time_diff, read_size, mean);
    done = true;
    in_file.close();
  }
};

class DataScheduler
{
  static const size_t BUFFER_COUNT = 100;
  static const size_t OUT_BUFFER_COUNT = 100;
  Reader *reader;
  uint processed_until = 0;

public:
  bool done = false;
  // must write to buffer in the given order(ascending block index)
  circular_buffer<Chunk, DataScheduler::OUT_BUFFER_COUNT> out_buffer;
  explicit DataScheduler(Reader *reader) : reader(reader){};

  void loop()
  {
    int current_stream = 0;
    while (!reader->done || !reader->value_buffer.empty())
    {
      while (reader->value_buffer.empty())
      {
      }

      // We use streams to synchronize executions callbacks in the GPU.
      Chunk *chunk = reader->value_buffer.get();
      Chunk *device_chunk;
      Chunk *device_output_chunk;

      cudaCheck(hipMalloc(&device_chunk, sizeof(Chunk)));
      cudaCheck(hipMalloc(&device_output_chunk, sizeof(Chunk)));
      cudaCheck(hipMemcpy(device_chunk, chunk, sizeof(Chunk), hipMemcpyHostToDevice));
      dim3 threads_per_block(KERNEL_BLOCK_SIZE);
      dim3 blocks_per_grid(BUFFER_SIZE / KERNEL_BLOCK_SIZE);

      printf("using %d threads per block\n", threads_per_block.x * threads_per_block.y);
      printf("using %d blocks per grid\n", blocks_per_grid.x * blocks_per_grid.y);

      kernel_matrix_mult<<<blocks_per_grid, threads_per_block>>>(device_output_chunk, device_chunk);

      Chunk output_chunk;
      // Copy result from device memory to the host memory
      cudaCheck(hipMemcpy(&output_chunk, device_output_chunk, sizeof(Chunk), hipMemcpyDeviceToHost));

      this->out_buffer.put(output_chunk);
      free(chunk);

      // Free arrays in device memory
      cudaCheck(hipFree(device_chunk));
      cudaCheck(hipFree(device_output_chunk));
    }
    done = true;
  }
};

// writes to a one-dimensional CSV file
class Writer
{
  static const size_t BUFFER_COUNT = 100;

public:
  bool done = false;
  DataScheduler *scheduler;
  Writer(DataScheduler *scheduler) : scheduler(scheduler){};
  float *write(std::string filename)
  {
    std::ofstream out_file(filename, std::ofstream::binary);
    if (!out_file)
    {
      std::cerr << "Failed opening file" << std::endl;
      exit(1);
    }

    auto start = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());
    int read_size = 0;
    // write column header
    out_file << "x" << std::endl;
    while (!scheduler->done || !scheduler->out_buffer.empty())
    {
      while (scheduler->out_buffer.empty())
      {
        if (scheduler->done)
        {
          goto loop_end;
        }
      }
      Chunk chunk = scheduler->out_buffer.get();
      for (size_t i = 0; i < BUFFER_SIZE; i++)
      {
        out_file << std::to_string(chunk.data[i]) << std::endl;
      }
      read_size += BUFFER_SIZE * sizeof(float);
    }
  loop_end:
    auto end = std::chrono::time_point_cast<std::chrono::milliseconds>(std::chrono::system_clock::now());
    auto time_diff = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
    // read_size in bytes, time_diff in milliseconds
    float mean = ((float)read_size / (float)time_diff);
    std::printf("WRITER:\nTempo: %dms\nLido: %d bytes\nMédia: %f bytes/ms\n\n", time_diff, read_size, mean);
    done = true;
    out_file.close();
  }
};

int main()
{
  std::cout << "Initializing Reader..." << std::endl;
  Reader reader;
  std::cout << "Initializing Scheduler..." << std::endl;
  DataScheduler scheduler(&reader);
  std::cout << "Initializing Writer..." << std::endl;
  Writer writer(&scheduler);

  std::thread reader_thread(&Reader::read, &reader, "in.csv");
  std::thread scheduler_thread(&DataScheduler::loop, &scheduler);
  std::thread writer_thread(&Writer::write, &writer, "out.csv");

  reader_thread.join();
  scheduler_thread.join();
  writer_thread.join();
}
